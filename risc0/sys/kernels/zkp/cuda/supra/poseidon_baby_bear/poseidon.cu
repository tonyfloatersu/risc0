#include "hip/hip_runtime.h"
#include "poseidon_constants.cuh"

#define CELLS 24
#define FULL_ROUNDS 8
#define HALF_FULL_ROUNDS 4
#define PARTIAL_ROUNDS 21
#define RATE 16
#define OUT 8

extern __shared__ fr_t _scratchpad[];

class scratchpad {
public:
  __device__ __forceinline__ fr_t& operator[](uint32_t i) const {
    return _scratchpad[32 * CELLS * (threadIdx.x / 32) + i * 32 + threadIdx.x % 32];
  }
};

__device__ __forceinline__ void add_round_constants_cells(fr_t cells[CELLS],
                                                          const fr_t round_constants[CELLS]) {
#pragma unroll
  for (uint32_t i = 0; i < CELLS; i++) {
    cells[i] += round_constants[i];
  }
}

__device__ __forceinline__ void pow7_cells(fr_t cells[CELLS]) {
#pragma unroll
  for (uint32_t i = 0; i < CELLS; i++) {
#ifdef __CUDA_ARCH__
    cells[i] ^= 7;
#endif
  }
}

__device__ __forceinline__ void matrix_mul(fr_t cells[CELLS],
                                           const fr_t matrix[CELLS][CELLS],
                                           uint32_t iters = CELLS,
                                           uint32_t iter_start = 0) {
  scratchpad sc;

#pragma unroll 1
  for (uint32_t i = iter_start; i < iter_start + iters; i++) {
#ifdef __CUDA_ARCH__
    fr_t sum = fr_t::dot_product<CELLS>(matrix[i], cells);

    sc[i] = sum;
#endif
  }

#pragma unroll
  for (uint32_t i = 0; i < CELLS; i++) {
    cells[i] = sc[i];
  }
}

__device__ __forceinline__ void full_round(fr_t cells[CELLS],
                                           const fr_t round_constants[CELLS],
                                           const fr_t matrix[CELLS][CELLS],
                                           uint32_t matrix_mul_iters = CELLS,
                                           uint32_t matrix_mul_iter_start = 0) {
  add_round_constants_cells(cells, round_constants);
  pow7_cells(cells);
  matrix_mul(cells, matrix, matrix_mul_iters, matrix_mul_iter_start);
}

__device__ __forceinline__ void
partial_round(fr_t cells[CELLS], fr_t round_constant, const fr_t* sparse_matrix) {
#ifdef __CUDA_ARCH__
  cells[0] ^= 7;
#endif
  fr_t p0 = cells[0] + round_constant;
  fr_t s0;
#ifdef __CUDA_ARCH__
  s0 = fr_t::dot_product<CELLS>(p0, fr_t(m00), &sparse_matrix[0], cells + 1);
#endif

#pragma unroll
  for (uint32_t j = 1; j < CELLS; j++) {
    cells[j] += p0 * sparse_matrix[CELLS - 1 + j - 1];
  }
  cells[0] = s0;
}

__device__ __forceinline__ void poseidon_hash(fr_t cells[CELLS],
                                              uint32_t final_matrix_mul_iters,
                                              uint32_t final_matrix_mul_iter_start) {
  const fr_t(*round_constants) = (decltype(round_constants))poseidon_rc_dev;
  const fr_t(*mds_matrix)[CELLS] = (decltype(mds_matrix))poseidon_mds_dev;
  const fr_t(*pre_sparse_matrix)[CELLS] = (decltype(pre_sparse_matrix))poseidon_ps_dev;
  const fr_t(*sparse_matrices)[2 * (CELLS - 1)] = (decltype(sparse_matrices))poseidon_sm_dev;

  uint32_t rc_counter = 0;

#pragma unroll 1
  for (uint32_t r = 0; r < HALF_FULL_ROUNDS; r++) {
    full_round(cells, &round_constants[rc_counter], r == 3 ? pre_sparse_matrix : mds_matrix);
    rc_counter += CELLS;
  }

  {
    add_round_constants_cells(cells, &round_constants[rc_counter]);
    rc_counter += CELLS;
  }

#pragma unroll 1
  for (uint32_t r = 0; r < PARTIAL_ROUNDS; r++) {
    partial_round(cells, round_constants[rc_counter], sparse_matrices[r]);
    rc_counter += 1;
  }

#pragma unroll 1
  for (uint32_t r = 0; r < HALF_FULL_ROUNDS; r++) {
    if (r == 3)
      full_round(cells,
                 &round_constants[rc_counter],
                 mds_matrix,
                 final_matrix_mul_iters,
                 final_matrix_mul_iter_start);
    else
      full_round(cells, &round_constants[rc_counter], mds_matrix);
    rc_counter += CELLS;
  }
}

__device__ __forceinline__ void poseidon_hash(uint32_t final_matrix_mul_iters,
                                              uint32_t final_matrix_mul_iter_start) {
  scratchpad sc;

  fr_t cells[CELLS];

  for (uint32_t i = 0; i < CELLS; i++) {
    cells[i] = sc[i];
  }

  poseidon_hash(cells, final_matrix_mul_iters, final_matrix_mul_iter_start);

  for (uint32_t i = 0; i < CELLS; i++) {
    sc[i] = cells[i];
  }
}

__launch_bounds__(256, 3) __global__ void _poseidon_fold(fr_t* out, const fr_t* in) {
  uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;

  fr_t cells[CELLS];

// set last OUT elements of CELLS to 0
#pragma unroll
  for (uint32_t i = RATE; i < CELLS; i++) {
    cells[i] = fr_t(0);
  }

  // load RATE elements into the start of CELLS
  const fr_t* in_off = in + tid * RATE;
#pragma unroll
  for (uint32_t i = 0; i < RATE; i++) {
    cells[i] = in_off[i];
  }

  poseidon_hash(cells, OUT, 0);

  // return first OUT elements of CELLS as output
  fr_t* inout_off = out + tid * OUT;
#pragma unroll
  for (uint32_t i = 0; i < OUT; i++) {
    inout_off[i] = cells[i];
  }
}

__launch_bounds__(256, 3) __global__
    void _poseidon_rows(fr_t* out, const fr_t* matrix, uint32_t count, uint32_t col_size) {
  uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;

  scratchpad sc;

// set CELLS to 0 initially
#pragma unroll
  for (uint32_t i = 0; i < CELLS; i++) {
    sc[i] = fr_t(0);
  }

  uint32_t body_iterations = (col_size / RATE) * RATE;
  uint32_t leftovers = col_size - body_iterations;
  if (leftovers == 0) {
    body_iterations -= RATE;
    leftovers = RATE;
  }
  for (uint32_t i = 0; i < body_iterations; i += RATE) {
    for (uint32_t j = 0; j < RATE; j++) {
      sc[j] += matrix[(i + j) * count + tid];
    }
    poseidon_hash(CELLS, 0);
  }
  if (leftovers > 0 || count == 0) {
    for (uint32_t i = 0; i < leftovers; i++) {
      sc[i] += matrix[(body_iterations + i) * count + tid];
    }
    poseidon_hash(OUT, 0);
  }

  // return first OUT elements of CELLS as output
  fr_t* inout_off = out + tid * OUT;
#pragma unroll
  for (uint32_t i = 0; i < OUT; i++) {
    inout_off[i] = sc[i];
  }
}
